#include <hip/hip_runtime.h>
#include <stdio.h>
#define MAX 1000
#define EPOCH 10

__global__ void addVector(int *c, int *a, int *b) {
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

int main() {
  int a[MAX], b[MAX], c[MAX];
  float time, time_avg = 0;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  for (int i = 0; i < MAX; i++) {
    a[i] = i;
    b[i] = i * i;
  }

  for (int t = 0; t < EPOCH; t++) {
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipMalloc((void **)&dev_a, MAX * sizeof(int));
    hipMalloc((void **)&dev_b, MAX * sizeof(int));
    hipMalloc((void **)&dev_c, MAX * sizeof(int));

    hipMemcpy(dev_a, a, MAX * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, MAX * sizeof(int), hipMemcpyHostToDevice);
    addVector<<<1, MAX>>>(dev_c, dev_a, dev_b);
    hipMemcpy(c, dev_c, MAX * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < MAX; i++) {
      if (i % 100 == 0) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
      }
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    time_avg += time;
    printf("\n\n");
  }

  printf("time : %f\n", time_avg / EPOCH);
  return 0;
}